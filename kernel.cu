
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    int row = blockIdx.y * blockDim.y +threadIdx.y;
    int col = blockIdx.y * blockDim.y +threadIdx.y;

    if (row < dim && col < dim) {
	int idx = row * dim + col;
	C[idx] = A[idx] + B[idx];
	}
} 
        

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    dim3 dim_block(BLOCK_SIZE);
    dim3 dim_grid((dim+BLOCK_SIZE-1) / BLOCK_SIZE, (dim+BLOCK_SIZE-1) / BLOCK_SIZE);

	// Invoke CUDA kernel -----------------------------------------------------
    matAdd<<<dim_grid, dim_block>>>(dim, A, B, C);

}

